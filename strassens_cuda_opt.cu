#include<hip/hip_runtime.h>
#include <stdlib.h>
#define idx(x,y,M) (M*(x)+(y))
using namespace std;

/* Performs strassens matrix multiplication using given matrices and their dimension.
 *
 * Returns a dynamically allocated flattened array for matrix containing result
 *
 * Arguments:
 * a,b: Matrices you want to multiple;
 * mult: Matrix you want to return results into;
 * int d11: Number of rows in a;
 * int d12: Number of columns in a, note that d12=d21 so dont need both;
 * int d22: Number of columns in b;
 * a * b = mult -> (d11,d12) * (d12,d22) = (d11,d22)
 */

__global__ void gpu_add(double *A, double *B, double *C,
        int idx_Ar, int idx_Ac, 
        int idx_Br, int idx_Bc, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < size) && (col  < size)){
        C[idx(row,col,size)] = A[idx(row+idx_Ar,col+idx_Ac,size)] + B[idx(row+idx_Br,col+idx_Bc,size)]; 
    }
}

__global__ void gpu_sub(double *A, double *B, double *C, 
        int idx_Ar, int idx_Ac,
        int idx_Br, int idx_Bc, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < size) && (col  < size)){
        C[idx(row,col,size)] = A[idx(row+idx_Ar,col+idx_Ac,size)] - B[idx(row+idx_Br,col+idx_Bc,size)]; 
    }
}

__global__ void gpu_ext(double *A, double *B, 
        double *C_A00, double *C_B00,
        double *C_A21, double *C_B21, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < size) && (col  < size)){
        int idx_r = 0; int idx_c = 0;
        C_A00[idx(row,col,size)] = A[idx(row+idx_r,col+idx_c,size)]; 
        C_B00[idx(row,col,size)] = B[idx(row+idx_r,col+idx_c,size)]; 

        idx_r = size;
        idx_c = 0;
        C_A21[idx(row,col,size)] = A[idx(row+idx_r,col+idx_c,size)]; 
        C_B21[idx(row,col,size)] = B[idx(row+idx_r,col+idx_c,size)]; 
    }
}

__global__ void mult_small(double *a, double *b, double *c,
        int idx_Ar, int idx_Ac, 
        int idx_Br, int idx_Bc, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < size) && (col  < size)){
        double value = 0;
        for(int k = 0; k < size; k++){
            value += a[idx(row+idx_Ar,col+idx_Ac+k,size)] * b[idx(idx_Br+row+k,idx_Bc+col,size)]; 
        }
        c[idx(row,col,size)] = value; 
    }
}

__global__ void gpu_synth(double *c11, double *c12, double *c21, double *c22, double* mult, int newSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int d11 = 2*newSize;

    if((row < newSize) && (col  < newSize)){
        for (int i = 0; i < newSize ; i++) {
            for (int j = 0 ; j < newSize ; j++) {
                mult[idx(i,j,d11)] = c11[idx(i,j,newSize)];
                mult[idx(i,(j + newSize),d11)] = c12[idx(i,j,newSize)];
                mult[idx((i + newSize),j,d11)] = c21[idx(i,j,newSize)];
                mult[idx((i + newSize),(j + newSize),d11)] = c22[idx(i,j,newSize)];
                //printf("%f ",mult[idx(i,j,d11)]);
            }
        }
    }
}

void strassen_multiply(double* A, double* B, double* mult, int d11) {
    if(d11 <= 32) {
        dim3 tbp(8,8);
        dim3 numBlocks((d11/tbp.x<1)? 1:d11/tbp.x, (d11/tbp.y<1)? 1:d11/tbp.y);

        mult_small <<< numBlocks, tbp >>> (A,B,mult,0,0,0,0,d11);
        return; 
    } else {
        int newSize = d11/2;
        dim3 tbp(8,8);
        dim3 numBlocks((newSize/tbp.x<1)? 1:newSize/tbp.x, (newSize/tbp.y<1)? 1:newSize/tbp.y);

        double a11r = 0;
        double a11c = 0;

        double a12r = 0;
        double a12c = newSize;

        double a21r = newSize;
        double a21c = 0;

        double a22r = newSize;
        double a22c = newSize;

        double b11r = 0;
        double b11c = 0;

        double b12r = 0;
        double b12c = newSize;

        double b21r = newSize;
        double b21c = 0;

        double b22r = newSize;
        double b22c = newSize;

        double *dc11, *dc12, *dc21, *dc22;
        hipMalloc((void**)&dc11,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dc12,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dc21,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dc22,(newSize*newSize)*sizeof(double));

        double *dA11, *dB11, *dA22, *dB22;
        hipMalloc((void**)&dB11,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dA11,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dA22,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dB22,(newSize*newSize)*sizeof(double));
        gpu_ext <<< numBlocks, tbp >>> (A, B, dA11, dB11, dA22, dB22, newSize);

        double *dS1, *dS2, *dS3, *dS4;
        hipMalloc((void**)&dS1,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dS2,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dS3,(newSize*newSize)*sizeof(double));
        hipMalloc((void**)&dS4,(newSize*newSize)*sizeof(double));

        // C11=s9*s10
        // s9 = a21 - a11
        // s10 = b11 + b12
        gpu_sub <<< numBlocks, tbp >>> (A, A, dS1, a21r, a21c, a11r, a11c, newSize);
        gpu_add <<< numBlocks, tbp >>> (B, B, dS2, b11r, b11c, b12r, b12c, newSize);
        strassen_multiply(dS1, dS2, dc22, newSize);

        // C22=s7*s8
        // s7 = a12 - a22
        // s8 = b21 + b22
        gpu_sub <<< numBlocks, tbp >>> (A, A, dS3, a12r, a12r, a22r, a22c, newSize);
        gpu_add <<< numBlocks, tbp >>> (B, B, dS4, b21r, b21c, b22r, b22c, newSize);
        strassen_multiply(dS3, dS4, dc11, newSize);

        // s3 = a21 + a22
        gpu_add <<< numBlocks, tbp >>> (A, A, dS1, a21r, a21c, a22r, a22c, newSize);
        strassen_multiply(dS1, dB11, dc21, newSize);
        gpu_sub <<< numBlocks, tbp >>> (dc22, dc21, dc22, 0, 0, 0, 0, newSize);

        // s2 = a11 + a12
        gpu_add <<< numBlocks, tbp >>> (A, A, dS3, a11r,a11c, a12r, a12c, newSize);
        strassen_multiply(dS3, dB22, dc12, newSize);
        gpu_sub <<< numBlocks, tbp >>> (dc11, dc12, dc11, 0, 0, 0, 0, newSize);



        // s1 = b12 - b22
        gpu_sub <<< numBlocks, tbp >>> (B, B, dS1, b12r, b12c, b22r, b22c, newSize);
        strassen_multiply(dA11, dS1, dS2, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc12, dS2, dc12, 0, 0, 0, 0, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc22, dS2, dc12, 0, 0, 0, 0, newSize);

        // s4 = b21 - b11
        gpu_sub <<< numBlocks, tbp >>> (B, B, dS3, b21r, b21c, b11r, b11c, newSize);
        strassen_multiply(dA22, dS3, dS4, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc11, dS4, dc11, 0, 0, 0, 0, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc21, dS4, dc21, 0, 0, 0, 0, newSize);
        
        // s5 = a11 + a22
        // s6 = b11 + b22
        gpu_add <<< numBlocks, tbp >>> (A, A, dS1, a11r, a11c, a22r, a22c, newSize);
        gpu_add <<< numBlocks, tbp >>> (B, B, dS2, b11r, b11c, b22r, b22c, newSize);
        strassen_multiply(dS1, dS2, dS3, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc11, dS3, dc11, 0, 0, 0, 0, newSize);
        gpu_add <<< numBlocks, tbp >>> (dc22, dS3, dc22, 0, 0, 0, 0, newSize);

        hipDeviceSynchronize();

        hipFree(dS1);
        hipFree(dS2);
        hipFree(dS3);
        hipFree(dS4);

        gpu_synth <<< numBlocks, tbp >>> (dc11,dc12,dc21,dc22,mult,newSize);

        hipFree(dc11);
        hipFree(dc12);
        hipFree(dc21);
        hipFree(dc22);

        return;
    }
}

void strassen_root(double* a, double* b, double* mult, int d11) {
    double *da, *db, *dc;

    hipMalloc((void**)&da,(d11*d11)*sizeof(double));
    hipMemcpy(da,a,(d11*d11)*(sizeof(double)),hipMemcpyHostToDevice);

    hipMalloc((void**)&db,(d11*d11)*sizeof(double));
    hipMemcpy(db,b,(d11*d11)*(sizeof(double)),hipMemcpyHostToDevice);

    hipMalloc((void**)&dc,(d11*d11)*sizeof(double));
    strassen_multiply(da,db,dc,d11);
    hipMemcpy(mult,dc,(d11*d11)*(sizeof(double)),hipMemcpyDeviceToHost);

}

void mat_multiply(double* a, double* b, double* mult, int d11, int d12, int d22) {
    // Initializing elements of matrix mult to 0.

    hipEvent_t time_s, time_e;
    hipEventCreate(&time_s);
    hipEventCreate(&time_e);

    hipEventRecord(time_s);
    strassen_root(a,b,mult,d11);
    hipEventRecord(time_e);
    hipEventSynchronize(time_e);

    float time = 0;
    hipEventElapsedTime(&time, time_s, time_e);

    printf("Time the function took is %.5fns", time);
    return;
}
