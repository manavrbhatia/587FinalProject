#include<hip/hip_runtime.h>

#define TILE_WIDTH 8
#define idx(x,y,M) (M*(x)+(y))

__global__ void naive_mult(double *A, double *B, double *C, int size){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if((row < size) && (col  < size)){
        double value = 0;
        for(int k = 0; k < size; k++){
            value += A[row*size + k] * B[k*size + col]; 
        }
        C[row*size + col] = value; 
    }
}

__global__ void naive_mult_tile(double *A, double *B, double *C, int size){
    __shared__  double subA[TILE_WIDTH][TILE_WIDTH];
    __shared__ double subB[TILE_WIDTH][TILE_WIDTH];
    
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;

    int row = blockRow*TILE_WIDTH + threadRow; 
    int col = blockCol*TILE_WIDTH + threadCol; 

    double value = 0;
    for(int sub_i = 0; sub_i < size/TILE_WIDTH; sub_i++) {
        subA[threadRow][threadCol] = A[idx(row, sub_i*TILE_WIDTH+threadCol, size)];
        subB[threadRow][threadCol] = B[idx(sub_i*TILE_WIDTH+threadRow, col, size)];
        __syncthreads();
   
        for(int k = 0; k < TILE_WIDTH; k++){
            value += subA[threadRow][k] * subB[k][threadCol]; 
        }
        __syncthreads();
    }
    C[row*size + col] = value; 
}

void mat_multiply(double* A, double* B, double* C, int d11, int d12, int d22) {
    // Initializing elements of matrix mult to 0.
    for(int i = 0; i < d11; ++i)
        for(int j = 0; j < d22; ++j)
            C[idx(i,j,d11)]=0;
    
    hipEvent_t time_s, time_e;
    hipEventCreate(&time_s);
    hipEventCreate(&time_e);

    double *dA, *dB, *dC;
    hipMalloc((void**)&dA,(d11*d12)*sizeof(double));
    hipMalloc((void**)&dB,(d12*d22)*sizeof(double));
    hipMalloc((void**)&dC,(d11*d22)*sizeof(double));

    hipMemcpy(dA,A,(d11*d12)*(sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(dB,B,(d12*d22)*(sizeof(double)),hipMemcpyHostToDevice);
    hipMemcpy(dC,C,(d11*d22)*(sizeof(double)),hipMemcpyHostToDevice);

    dim3 tbp(8,8);
    dim3 numBlocks((d11/tbp.x<1)? 1:d11/tbp.x, (d11/tbp.y<1)? 1:d11/tbp.y);

    hipEventRecord(time_s);
    naive_mult_tile <<< numBlocks, tbp >>> (dA, dB, dC, d12);
    hipEventRecord(time_e);
    hipEventSynchronize(time_e);

    float time = 0;
    hipEventElapsedTime(&time, time_s, time_e);
    printf("Time the function took is %.5fns", time);
    hipMemcpy(C,dC,(d11*d22)*(sizeof(double)),hipMemcpyDeviceToHost);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return;
}
